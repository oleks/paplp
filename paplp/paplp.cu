#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cstddef>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "types.hpp"
#include "problem.hpp"

using std::cout;
using std::endl;

using std::string;

texture<float, hipTextureType2D, hipReadModeElementType> textureRef;

void randomFill(Data* values, int length, Data rand_max)
{
  Data quotient = ((Data)RAND_MAX) / rand_max;
  for ( size_t i = 0; i < length; ++i )
  {
    values[i] = ((Data)rand()) / quotient;
  }
}

template<size_t size, typename type>
void print(type const(& array)[size])
{
  if ( size == 0 )
  {
    cout << "[]" << endl;
    return;
  }

  cout << '[';
  size_t lastIndex = size - 1;
  for ( size_t i = 0 ; i < lastIndex ; ++i )
  {
    cout << array[i] << ", ";
  }
  cout << array[lastIndex] << ']' << endl;
}

int main()
{

  Data e = 0.1;
  size_t n = 3;
  size_t m = 3;
  Data d[3] = { 7, 8, 9 };
  Data b[3] = { 4, 5, 6 };
  Data c[9] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };

  Problem initialProblem(e, n, m, d, c, b);
  SpecialProblem specialProblem = (SpecialProblem)initialProblem;

  printf("%e\n", specialProblem.Epsilon());

/*
  print(c);

  convertToSpecialForm(&initialProblem, &specialProblem);
  ensureSpecialForm(&specialProblem);

  print(c);
*/
/*
  int N = 200;
  int BLOCK = 10;
  int SIZE = sizeof(data) * N;

  data* hostValues = (data*)malloc(SIZE);
  randomFill(hostValues, N, 100.0f);

  data* deviceValues;
  hipMalloc(&deviceValues, SIZE);

  hipMemcpy(deviceValues, hostValues, SIZE, hipMemcpyHostToDevice);
  exponent<<<N/BLOCK, BLOCK>>>(deviceValues);
  hipMemcpy(hostValues, deviceValues, SIZE, hipMemcpyDeviceToHost);

*/

  return 0;
}
